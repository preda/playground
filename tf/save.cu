
#include <hip/hip_runtime.h>
/*
struct Test { unsigned p; u64 k; };

#include "tests.inc"

__global__ void test(unsigned p, u64 k) {
  U3 q = makeQ(p, k);
  testOut = expMod(p, q);
}

static void selfTest() {
  int n = sizeof(tests) / sizeof(tests[0]);
  for (Test *t = tests, *end = tests + n; t < end; ++t) {
    unsigned p = t->p;
    u64 k = t->k;
    int shift = __builtin_clz(p);
    assert(shift < 27);
    p <<= shift;
    // printf("p %u k %llu m: ", t->p, t->k); print(m);
    test<<<1, 1>>>(p, k);
    cudaDeviceSynchronize();
    cudaError_t err = cudaGetLastError();
    if (err != cudaSuccess) {
      printf("CUDA error: %s\n", cudaGetErrorString(err));
      break;
    } else {
      if (testOut.a != 1 || testOut.b || testOut.c) {
        printf("ERROR %10u %20llu m ", t->p, t->k); print(m); print(out);
        break;
      } else {
        // printf("OK\n");
      }
    }
  }
}
*/

/*
bool launch(unsigned p, u64 k0, int t, unsigned *classes, int repeat) {
  cudaDeviceSynchronize();
  cudaError_t err = cudaGetLastError();
  if (err != cudaSuccess) {
    printf("CUDA error: %s\n", cudaGetErrorString(err));
    return true;
  }
  if (foundFactor) {
    printf("factor %llu\n", foundFactor);
    return true;
  }
  memcpy(classTab, classes, t * sizeof(unsigned));
  if (t < THREADS_PER_GRID) {
    printf("Tail %d\n", t);
    memset(classTab + t, 0xff, (THREADS_PER_GRID - t) * sizeof(unsigned));
  }
  tf<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(p, k0, k0);
  return false;
}

int findFactor(unsigned p, u64 k0, int repeat) {
  u64 timeStart = timeMillis();
  u64 time1 = timeStart;
  unsigned classes[THREADS_PER_GRID];
  int accepted = 0;
  int t = 0;
  int c = 0;
  int nLaunch = 0;
  for (; c < NCLASS; ++c) {
    if (acceptClass(p, c)) {
      classes[t++] = c;
      if (t >= THREADS_PER_GRID) {
        accepted += THREADS_PER_GRID;
        t = 0;
        ++nLaunch;
        if (launch(p, k0, THREADS_PER_GRID, classes, repeat)) { return -1; }
        if (!(nLaunch & 0xf)) {
          u64 time2 = timeMillis();
          printf("%8u: %u ms\n", c, (unsigned)(time2 - time1));
          time1 = time2;
        }
      }
    }
  }
  accepted += t;
  launch(p, k0, t, classes, repeat);
  u64 time2 = timeMillis(); time1 = time2;
  printf("%8u: %u ms; total %llu\n", c, (unsigned)(time2 - time1), time2 - timeStart);
  return accepted;
}
*/

    /*    
    u32 *pw = words + tid;
    int popCount = 0;
    for (u32 *p = words + tid, *end = words + NWORDS; p < end; p += THREADS_PER_BLOCK) {
      u32 w = ~*p;
      *p = w;
      popCount += __popc(w);
    }
    __syncthreads();
    u32 save = *words;
    u32 *countp = words;
    u32 *outPos = words + atomicAdd(countp, popCount);
    __syncthreads();
    *words = save;
    for (u32 *p = words + tid, *end = words + NWORDS; p < end; p += THREADS_PER_BLOCK) {
      u32 w = *p;
      extractBits(out, *p, ((p - words) << 5));
    }
    __syncthreads();
  }
  
      while (bits) {
        int bit = bfind(bits);
        bits &= ~(1<<bit);
        *outPos++ = bitBase + bit;
      }
      */  
  
  /*
  unsigned c = classTab[id];
  if (c == 0xffffffff) { return; }
  u64 k = k0 + c;
  for (int i = repeat; i > 0; --i) {
    if (isFactor(exp, k)) {
      printf("%d found factor %llu\n", id, k);
      deviceFactor = k;
      break;
    }
    k += NCLASS;
  }
  */


/*
__device__ int bumpBtc(int btc, u64 delta, u16 prime) {
  return ((btc -= delta % prime) < 0) ? (btc + prime) : btc; 
}

#define REPEAT_32(w, s) w(11)s w(13)s w(17)s w(19)s w(23)s w(29)s w(31)
#define REPEAT_64(w, s) w(37)s w(41)s w(43)s w(47)s w(53)s w(59)s w(61)
#define REPEAT(w, s) REPEAT_32(w, s)s REPEAT_64(w, s)
*/

/*
__device__ u16 invTab[ASIZE(primes)];
__global__ void initInvTab(u64 step) {
  int id = ID;
  u16 prime = primes[id];
  u16 inv = modInv16(step, prime);
  assert(inv == modInv32(step, prime));
  invTab[id] = inv;
}
*/

/*
__device__ u16 modInv16(u64 step, u16 prime) {
  u16 n = step % prime;
  u16 q = prime / n;
  u16 d = prime - q * n;
  int x = -q;
  int prevX = 1;
  while (d) {
    q = n / d;
    { u16 save = d; d = n - q * d; n = save;         }
    { int save = x; x = prevX - q * x; prevX = save; }
  }
  return (prevX >= 0) ? prevX : (prevX + prime);
}

__device__ u16 bitToClear(u32 exp, u64 k, u16 prime) {
  u64 step = 2 * NCLASS * (u64) exp;  
  u16 inv = modInv16(step, prime);
  assert(inv == modInv32(step, prime));
  return bitToClear(exp, k, prime, inv);
}

__device__ u16 bitToClear(u32 exp, u64 k, u16 prime, u16 inv) {
  u16 kmod = k % prime;
  u16 qmod = ((exp << 1) * (u64) kmod + 1) % prime;
  return (prime - qmod) * (u32) inv % prime;
}

__device__ u16 classBtc(u32 exp, u16 c, u16 prime, u16 inv) {
  u16 qInv = (c * (u64) (exp << 1) + 1) * inv % prime;
  u16 btc = qInv ? (prime - qInv) : qInv;
  assert(btc == bitToClear(exp, c, prime, inv));
  return btc;
}
*/
