#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

typedef unsigned long long u64;

typedef struct {
  unsigned d0, d1, d2;
} N96;

typedef struct {
  unsigned d0, d1, d2, d3, d4, d5;
} N192;

typedef struct {
  unsigned d0, d1, d2, d3;
} N128;

__device__ static unsigned mul(unsigned a, unsigned b) {
  return a * b;
}

__device__ static unsigned madhi_cc(unsigned a, unsigned b, unsigned c) {
  unsigned r;
  asm("mad.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ static unsigned madhic_cc(unsigned a, unsigned b, unsigned c) {
  unsigned r;
  asm("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
  return r;
}

__device__ static unsigned mulhic(unsigned a, unsigned b) {
  unsigned r;
  asm("madc.hi.u32 %0, %1, %2, 0;" : "=r"(r) : "r"(a), "r"(b));
  return r;
}

__device__ static unsigned add_cc(unsigned a, unsigned b) {
  unsigned r;
  asm("add.cc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned addc_cc(unsigned a, unsigned b) {
  unsigned r;
  asm("addc.cc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned addc(unsigned a, unsigned b) {
  unsigned r;
  asm("addc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned sub_cc(unsigned a, unsigned b) {
  unsigned r;
  asm("sub.cc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned subc_cc(unsigned a, unsigned b) {
  unsigned r;
  asm("subc.cc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned subc(unsigned a, unsigned b) {
  unsigned r;
  asm("subc.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
}

__device__ static unsigned shfl(unsigned a, unsigned b, unsigned n) {
  unsigned r;
  asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(n));
  return r;
}

// 4W = 3W * 1W 
__device__ static N128 mul(N96 a, unsigned n) {
  N128 r;
  r.d0 =         mul(n, a.d0);
  r.d1 =  add_cc(mulhi(n, a.d0), mul(n, a.d1));
  r.d2 = addc_cc(mulhi(n, a.d1), mul(n, a.d2));
  r.d3 =    addc(mulhi(n, a.d2), 0);
  return r;
}

// 6W = 6W - 6W
__device__ static N192 subshl(N192 a, N192 b, int n) {
  N192 r;
  r.d0 =  sub_cc(a.d0, b.d0 << n);
  r.d1 = subc_cc(a.d1, shfl(b.d0, b.d1, n));
  r.d2 = subc_cc(a.d2, shfl(b.d1, b.d2, n));
  r.d3 = subc_cc(a.d3, shfl(b.d2, b.d3, n));
  r.d4 = subc_cc(a.d4, shfl(b.d3, b.d4, n));
  r.d5 =    subc(a.d5, shfl(b.d4, b.d5, n));
  return r;
}

__device__ static N192 shl64(N128 a) {
  N192 r = {0, 0, a.d0, a.d1, a.d2, a.d3};
  return r;
}

__device__ static N192 shl32(N128 a) {
  N192 r = {0, a.d0, a.d1, a.d2, a.d3, 0};
  return r;
}

__device__ static N192 shl0(N128 a) {
  N192 r = {a.d0, a.d1, a.d2, a.d3, 0, 0};
  return r;
}

// 3W = 6W % 3W; b >= 2**95
__device__ static N96 mod(N192 a, N96 b) {
  assert(b.d2 & 0x80000000);
  const u64 R64 = 0xffffffffffffffffULL / ((0x100000000ULL | shfl(b.d1, b.d2, 1)) + 1); // (((u64) b.d2) << 1) | (b.d1 >> 31));
  assert((R64 >> 32) == 0);
  const unsigned R = (unsigned) R64;
  unsigned n;
  N192 c;

  unsigned saveA5 = a.d5;
  
  n = mulhi(a.d5, R);
  c = shl64(mul(b, n));
  a = subshl(a, c, 1);
  if (a.d5 & 0xfffffff0) {
    printf("a5 %08x b2 %08x b1 %08x R %08x n %08x rem %08x\n", saveA5, b.d2, b.d1, R, n, a.d5);
  }
  assert((a.d5 & 0xfffffff0) == 0);

  n = mulhi(shfl(a.d4, a.d5, 28), R);
  c = shl32(mul(b, n));
  a = subshl(a, c, 5);
  assert(a.d5 == 0 && (a.d4 & 0xffffff00) == 0);

  n = mulhi(shfl(a.d3, a.d4, 24), R);
  c = shl0(mul(b, n));
  a = subshl(a, c, 9);
  assert(a.d5 == 0 && a.d4 == 0 && (a.d3 & 0xfffff000) == 0);

  n = mulhi(shfl(a.d2, a.d3, 20), R) >> 19;
  c = shl0(mul(b, n));
  a = subshl(a, c, 0);
  assert(a.d5 == 0 && a.d4 == 0 && a.d3 == 0);
  
  N96 ret = {a.d0, a.d1, a.d2};
  return ret;
}

__global__ void testMod(N96 *out, N192 *as, N96 *bs) {
  // printf("dim %d\n", blockDim.y);
  int x = threadIdx.x;
  int y = threadIdx.y;
  N192 a = as[x];
  N96 b = bs[y];
  out[x + y * blockDim.y] = mod(a, b);
}

unsigned random32() {
  return (((unsigned)random()) << 1) | (random() & 1);
}

N96 random96() {
  N96 ret = {random32(), random32(), random32()};
  return ret;
}

N192 random192() {
  N192 ret = {random32(), random32(), random32(), random32(), random32(), random32()};
  return ret;
}

#define N 32

__managed__ N192 as[N];
__managed__ N96 bs[N];
__managed__ N96 out[N * N];

int main() {
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  for (int i = 0; i < N; ++i) {
    N192 a = random192();
    a.d5 |= 0x80000000;
    as[i] = a;
  }

  for (int i = 0; i < N; ++i) {
    N96 b = random96();
    b.d2 |= 0x80000000;
    bs[i] = b;
  }
  printf("Done rnd\n");

  dim3 blockDim(N, N);
  testMod<<<1, blockDim>>>(out, as, bs);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}



/*
__global__ void testMul2(unsigned *out, unsigned d0, unsigned d1, unsigned d2, unsigned n) {
  unsigned r;
  asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(r) : "r"(d0), "r"(d1), "r"(n));
  *out = r;
  // int id = threadIdx.x;
  // mul96a(out, d0, d1, d2, n);
  // printf("%u: %x %x %x %x\n", n, r[0], r[1], r[2], r[3]);
}
*/

/*
// 6W = 6W - 6W
__device__ static void sub192(unsigned *out, unsigned *a, unsigned *b) {
  out[0] = sub_cc(a[0], b[0]);
  out[1] = subc_cc(a[1], b[1]);
  out[2] = subc_cc(a[2], b[2]);
  out[3] = subc_cc(a[3], b[3]);
  out[4] = subc_cc(a[4], b[4]);
  out[5] = subc(a[5], b[5]);
}
*/

/*
__device__ static void mul96a(unsigned *out, unsigned d0, unsigned d1, unsigned d2, unsigned n) {
  out[0] = mul(n, d0);
  out[1] = madhi_cc(n, d0, mul(n, d1));
  out[2] = madhic_cc(n, d1, mul(n, d2));
  out[3] = mulhic(n, d2);
}
*/

/*
__global__ void AAA(unsigned long long *out, unsigned long long a, unsigned long long b) {
  *out = a * b;
}

__global__ void BBB(unsigned *out, unsigned a, unsigned b, unsigned c) {
  unsigned r;
  asm("madc.lo.u32 %0, %1, %2, %3;": "=r"(r): "r"(a), "r"(b), "r"(c));
  *out = r;
}
*/


// #define mul(a, b) ((a) * (b))
  /*
  unsigned r;
  asm("mul.lo.u32 %0, %1, %2;" : "=r" (r) : "r" (a), "r" (b));
  return r;
  */
