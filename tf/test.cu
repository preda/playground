#include "hip/hip_runtime.h"
// Copyright (c) Mihai Preda, 2015.

/*
  This is a program for trial factoring of Mersenne numbers, which are numbers of the form
  2^exp - 1. See http://www.mersenne.org/various/math.php

  For a given mersenne number 2^exp-1, i.e. for a given value "exp", exp being prime,
  the trial factors are of the form q = 2*k*exp + 1, and we're interested only in prime factors.

  Range: exp < 2^30. q < 2^94.

  A first step consist in generating prime candidate factors -- this is called "sieving" because
  it uses Erathostene's sieve. Next each factor q is tested by computing "modular exponentiation",
  reminder r = 2^exp modulo q. If this reminder is equal to 1, it means that q is a factor of
  2^exp-1, and thus the mersenne number is not prime.

  Both the sieving and the testing is run on the GPU.

  Some naming conventions used:
  shl: shift left.
  shr: shift right.
  U3: unsigned int using 3 words (i.e. 96bits).
  u32: unsigned int using 32 bits.
 */

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include "widemath.h"

#define ASIZE(a) (sizeof(a) / sizeof(a[0]))

// Threads per block, doing sieving and factor-testing.
#define THREADS_PER_BLOCK (512)

// How many words of shared memory to use for sieving.
#define NWORDS (6 * 1024)

// Must update acceptClass() when changing these.
#define NCLASS     (4 * 3 * 5 * 7 * 11)
// Out of NCLASS, how many classes pass acceptClass(). Sync with NCLASS.
#define NGOODCLASS (2 * 2 * 4 * 6 * 10)

// Returns whether 2 * c * exp + 1 is 1 or 7 modulo 8.
// Any Marsenne factor must be of this form. See http://www.mersenne.org/various/math.php
bool q1or7mod8(u32 exp, u32 c) {
  return !(c & 3) || ((c & 3) + (exp & 3) == 4);
}

// whether 2 * c * exp + 1 != 0 modulo prime
bool notMultiple(u32 exp, u32 c, unsigned prime) { return (2 * c * (u64) exp + 1) % prime; }

bool acceptClass(u32 exp, u32 c) {
#define P(p) notMultiple(exp, c, p)
  return q1or7mod8(exp, c) && P(3) && P(5) && P(7) && P(11);
#undef P
}

// Bits for sieving.
#define NBITS (NWORDS << 5)
// Number of pre-computed primes for sieving.
#define NPRIMES (ASIZE(primes))

u64 timeMillis() {
  struct timeval tv;
  gettimeofday(&tv, 0);
  return tv.tv_sec * 1000 + tv.tv_usec / 1000;
}

__device__ const u32 primes[] = {
#include "primes-1M.inc"
};

__managed__ u64 foundFactor;  // If a factor k is found, save it here.
__device__ u32 invTab[NPRIMES];
__device__ int btcTab[NPRIMES];

// returns (x*n >> 32) + (n ? 1 : 0). Used for Montgomery reduction. 5 MULs.
__device__ U3 mulM(U3 x, u32 n) {
  u32 a, b, c;
  asm("add.cc.u32     %0, 0xffffffff, %6;" // set carry = n
      "mul.hi.u32     %0, %3, %6;"
      "mul.lo.u32     %1, %5, %6;"
      "madc.lo.cc.u32 %0, %4, %6, %0;"
      "madc.hi.cc.u32 %1, %4, %6, %1;"
      "madc.hi.u32    %2, %5, %6, 0;"
      : "=r"(a), "=r"(b), "=r"(c)
      : "r"(x.a), "r"(x.b), "r"(x.c), "r"(n));
  return (U3) {a, b, c};
}

// returns x * x; 6 MULs.
__device__ U4 square(U2 x) {
  u32 a, b, c, d;
  asm(
      "mul.lo.u32     %1, %4, %5;"
      "mul.hi.u32     %2, %4, %5;"
      "mul.lo.u32     %0, %4, %4;"
      "add.cc.u32     %1, %1, %1;"
      "addc.cc.u32    %2, %2, %2;"
      "addc.u32       %3, 0, 0;"
      
      "mad.hi.cc.u32  %1, %4, %4, %1;"
      "madc.lo.cc.u32 %2, %5, %5, %2;"
      "madc.hi.u32    %3, %5, %5, %3;"
      : "=r"(a), "=r"(b), "=r"(c), "=r"(d)
      : "r"(x.a), "r"(x.b));
  return (U4) {a, b, c, d};
}

// returns x * x; 12 MULs.
__device__ U6 square(U3 x) {
  U2 ab = {x.a, x.b};
  U4 ab2 = square(ab);
  // U3 abc = mul(ab, x.c + x.c);
  U3 abc = ab * (x.c + x.c);
  
  u32 c, d, e, f;
  asm(
      "add.cc.u32  %0, %4, %6;"
      "addc.cc.u32 %1, %5, %7;"
      "mul.hi.u32  %3, %9, %9;"
      "madc.lo.cc.u32 %2, %9, %9, %8;"
      "addc.u32       %3, %3, 0;"
      : "=r"(c), "=r"(d), "=r"(e), "=r"(f)
      : "r"(ab2.c), "r"(ab2.d), "r"(abc.a), "r"(abc.b), "r"(abc.c), "r"(x.c));
  assert(!(f & 0xc0000000));
  return (U6) {ab2.a, ab2.b, c, d, e, f};
}

__device__ U5 modStep(U5 t, U3 m, u32 R, int sh, int bits) {
  u32 n = mulhi(shl(t.c, t.d, 32 - bits), R);
  t = (U5){0, t.a, t.b, t.c, t.d} - (_U5(m * n) << (sh + bits));
  assert(!t.e && !(t.d & (0xfffffff8 << bits)));
  return t;
}

__device__ U3 modShl3w(U4 x, U3 m) {
  assert(m.c && !(m.c & 0xc0000000));
  int sh = __clz(m.c) + 1;
  if (sh > 20) {
    m = m << (sh - 20);
    sh = 20;
  }
  assert(sh >= 3 && sh <= 20);
  u32 R = 0xffffffffffffffffULL / ((0x100000000ULL | shl(m.b, m.c, sh)) + 1);
  
  u32 n = mulhi(x.d, R);
  x -= (m * n) << sh;
  assert(!(x.d & 0xfffffff8));
  U5 t = _U5(x);

  t = modStep(t, m, R, sh, 3);
  t = modStep(t, m, R, sh, 6);
  t = modStep(t, m, R, sh, 9);
  
  n = mulhi(shl(t.c, t.d, 20), R) >> (20 - sh);
  x = (U4){t.a, t.b, t.c, t.d} - m * n;
  assert(!x.d && !(x.c >> (35 - sh)));
  return (U3) {x.a, x.b, x.c};
}

// Compute m' such that: (u32) (m * m') == 0xffffffff, using extended binary euclidian algorithm.
// See http://www.ucl.ac.uk/~ucahcjm/combopt/ext_gcd_python_programs.pdf
// m is odd.
__device__ static u32 mprime(u32 m) {
  m = (m >> 1) + 1;
  u32 u = m;
  u32 v = m << 31; 
  for (int i = 0; i < 30; ++i) {
    u = (u >> 1) + ((u & 1) ? m : 0);
    v = shr(v, u, 1);
  }
  return v | 1;
}

// Montgomery Reduction. 18 MULs.
// See https://www.cosic.esat.kuleuven.be/publications/article-144.pdf
// Returns x * U^-1 mod m
__device__ static U3 montRed(U6 x6, U3 m, u32 mp) {
  assert(!(x6.f & 0xc0000000));
  assert(x6.a + (x6.a * mp * m.a) == 0);
  U5 x5 = shr1w(x6) + mulM(m, x6.a * mp);
  U4 x4 = shr1w(x5) + mulM(m, x5.a * mp);
  U3 x3 = shr1w(x4) + mulM(m, x4.a * mp);
  assert(!(x3.c & 0xc0000000));
  return x3;
}

// returns 2^exp % m
__device__ U3 expMod(u32 exp, U3 m) {
  assert(exp & 0x80000000);
  int sh = exp >> 25;
  assert(sh >= 64 && sh < 128);
  U3 a = modShl3w((U4){0, 0, 1 << (sh - 64), 1 << (sh - 96)}, m);
  u32 mp = mprime(m.a);
  for (exp <<= 7; exp; exp += exp) {
    a = montRed(square(a), m, mp);
    if (exp & 0x80000000) { a <<= 1; }  // Alternative: a <<= exp >> 31;
  }
  return montRed(_U6(a), m, mp);
}

// returns whether (2*k*p + 1) is a factor of (2^p - 1)
__device__ bool isFactor(u32 exp, u32 flushedExp, u64 k) {
  U3 q = _U2(k) * (exp + exp) + (U3){1, 0, 0};  // 2 * k * exp + 1 as U3
  U3 r = expMod(flushedExp, q);
  return r.a == 1 && !r.b && !r.c;  
}

__device__ u32 modInv32(u64 step, u32 prime) {
  int n = step % prime;
  int q = prime / n;
  int d = prime - q * n;
  int x = -q;
  int prevX = 1;
  while (d) {
    q = n / d;
    { int save = d; d = n - q * d; n = save; }            // n = set(d, n - q * d);
    { int save = x; x = prevX - q * x; prevX = save; }    // prevX = set(x, prevX - q * x);
  }
  return (prevX >= 0) ? prevX : (prevX + prime);
}

// 3 times 64bit modulo, expensive!
__device__ int bitToClear(u32 exp, u64 k, u32 prime, u32 inv) {
  u32 kmod = k % prime;
  u32 qmod = (kmod * (u64) (exp << 1) + 1) % prime;
  return (prime - qmod) * (u64) inv % prime;
}

__global__ void __launch_bounds__(1024) initInvTab(u32 exp) {
  assert(gridDim.x * blockDim.x == NPRIMES);
  u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  invTab[id] = modInv32(2 * NCLASS * (u64) exp, primes[id]);
}

__global__ void __launch_bounds__(1024) initBtcTab(u32 exp, u64 k) {
  assert(gridDim.x * blockDim.x == NPRIMES);
  u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  btcTab[id] = bitToClear(exp, k, primes[id], invTab[id]);
}

// Returns the position of the most significant bit that is set.
__device__ int bfind(u32 x) { int r; asm("bfind.u32 %0, %1;": "=r"(r): "r"(x)); return r; }

/*extern __shared__ u32 words[];
//__noinline__
__device__ void sieve(int prime, int btc0) {
  int btcAux = btc0 - (int) (NCLASS * (u64) NBITS * blockIdx.x % prime);
  int btc = (btcAux < 0) ? btcAux + prime : btcAux;
  while (btc < NBITS) {
    atomicOr(words + (btc >> 5), 1 << (btc & 0x1f));
    btc += prime;
  }
}
*/

// #define TID (threadIdx.x)

__global__ void __launch_bounds__(THREADS_PER_BLOCK, 4) tf(u32 exp, u32 flushedExp, u64 k) {
  __shared__ u32 words[NWORDS];
  const int tid = threadIdx.x;
  for (int i = 0; i < NWORDS / THREADS_PER_BLOCK; ++i) { words[tid + i * THREADS_PER_BLOCK] = 0; }
  __syncthreads();
  // u64 delta = NCLASS * (u64) NBITS * blockIdx.x;
  // #pragma unroll
  int prime = primes[tid];
  int btc0  = btcTab[tid];
  for (int i = 1; i < NPRIMES / THREADS_PER_BLOCK + 1; ++i) {
    int btcAux = btc0 - (NCLASS * NBITS % prime) * blockIdx.x % prime;
    btc0  = btcTab[tid + i * THREADS_PER_BLOCK]; 
    int btc = (btcAux < 0) ? btcAux + prime : btcAux;
    while (btc < NBITS) {
      atomicOr(words + (btc >> 5), 1 << (btc & 0x1f));
      // atomicAnd(words + (btc >> 5), ~(1 << (btc & 0x1f)));
      btc += prime;
    }
    prime = primes[tid + i * THREADS_PER_BLOCK];
  }
  /*
  for (int i = tid; i < NPRIMES; i += THREADS_PER_BLOCK) {
    int prime = primes[i];
    int btcAux = btcTab[i] - (int) (delta % prime);
    int btc = (btcAux < 0) ? btcAux + prime : btcAux;
    while (btc < NBITS) {
      atomicOr(words + (btc >> 5), 1 << (btc & 0x1f));
      btc += prime;
    }
  }
  */
  __syncthreads();
  /*
  for (int i = 0; i < NWORDS / THREADS_PER_BLOCK; ++i) {
    words[tid + i * THREADS_PER_BLOCK] = ~words[tid + i * THREADS_PER_BLOCK];
  }
  __syncthreads();
  */

  k += (tid + blockIdx.x * NWORDS) * (u64) (32 * NCLASS);
  int i = tid;
  u32 bits = ~words[i];
  while (true) {
    while (!bits) {
      i += THREADS_PER_BLOCK;
      if (i >= NWORDS) { return; }
      bits = ~words[i];
      k += THREADS_PER_BLOCK * 32 * NCLASS;
    }
    int bit = bfind(bits);
    bits &= ~(1 << bit);
    if (isFactor(exp, flushedExp, k + bit * NCLASS)) {
      foundFactor = k + bit * NCLASS;      
    }
  }
}

int classTab[NGOODCLASS];

void initClasses(u32 exp) {
  int nClass = 0;
  for (int c = 0; c < NCLASS; ++c) {
    if (acceptClass(exp, c)) {
      classTab[nClass++] = c;
    }
  }
  assert(nClass == NGOODCLASS);
}

u64 calculateK(u32 exp, int bits) {
  return (((u128) 1) << (bits - 1)) / exp;
  // return k - k % NCLASS;
}

#define CUDA_CHECK_ERR  err = hipGetLastError(); if (err) { printf("CUDA error: %s\n", hipGetErrorString(err)); return 0; }

int main() {
  assert(NPRIMES % 1024 == 0);
  
  hipError_t err;
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  // hipSetDevice(1);
  CUDA_CHECK_ERR;
  
  const u32 exp = 119904229;
  u64 t1 = timeMillis();
  u64 t0 = t1;
  initClasses(exp);
  printf("initClasses: %llu ms\n", timeMillis() - t1);

  int startPow2 = 69;
  u64 kStart = calculateK(exp, startPow2);
  u64 kEnd   = calculateK(exp, startPow2 + 1);
  u64 k0Start = kStart - (kStart % NCLASS);
  u64 k0End   = kEnd + (NCLASS - (kEnd % NCLASS)) % NCLASS;
  u64 perRound = NBITS * (u64) NCLASS;
  u64 rounds = (kEnd - k0Start + (perRound - 1)) / perRound;
  u32 flushedExp = exp << __builtin_clz(exp);
  
  printf("exp %u kStart %llu kEnd %llu k0Start %llu k0End %llu, %llu rounds %llu\n",
         exp, kStart, kEnd, k0Start, k0End, (k0Start + rounds * perRound), rounds);
  
  t1 = timeMillis();
  initInvTab<<<NPRIMES/1024, 1024>>>(exp);
  hipDeviceSynchronize();
  printf("initInvTab: %llu ms\n", timeMillis() - t1);
  
  t1 = timeMillis();
  for (int cid = 0; cid < NGOODCLASS; ++cid) {
    int c = classTab[cid];
    u64 k = k0Start + c;
    initBtcTab<<<NPRIMES/1024, 1024>>>(exp, k);
    hipDeviceSynchronize();
    if (foundFactor) { printf("Factor K: %llu\n", foundFactor); break; }
    if (!(cid & 0xf)) {
      u64 t2 = timeMillis();
      printf("%5d: class %5d: %llu\n", cid, c, t2 - t1);
      t1 = t2;
    }
    tf<<<32 * 4 * 12 * 1024 / NWORDS, THREADS_PER_BLOCK/*, NWORDS * 4*/>>>(exp, flushedExp, k);
    // hipDeviceSynchronize(); CUDA_CHECK_ERR;

  }
  printf("Total time: %llu ms\n", timeMillis() - t0);
  // hipDeviceReset();
}


/*
    // u64 kWord = k0 + c + (tid + round * NWORDS) * (32 * NCLASS);
    // tid * (32 * NCLASS) + round * NCLASS * (u64) NBITS;

__global__ void test(u32 exp, U3 m) {
  u32 flushedExp = exp << __clz(exp);
  isFactor(flushedExp, m);
}

__global__ void __launch_bounds__(SIEVE_THREADS, 2) sieve(u32 exp, u64 k0, int c0) {
  __shared__ u32 words[NWORDS];
  const int tid = threadIdx.x;
  const int cid = c0 + blockIdx.x;
  const int c = classTab[cid];
  const u32 flushedExp = exp << __clz(exp);
  u64 kBlock = k0 + c + (tid - SIEVE_THREADS) * (32 * NCLASS) - NBITS * (u64) NCLASS;
  bool shouldExit = false;
  // u32 timeSieve, timeTest, timeCopy;

  for (int round = 0; round < nRounds && !shouldExit; ++round) {
    // u32 time0 = clock();
    if (tid < SIEVE_THREADS) {
      // u32 *btcp = btcTab + tid;
      for (const u32 *p = primes + tid, *end = primes + NPRIMES, *invp = invTab; p < end;
           p += SIEVE_THREADS, invp += SIEVE_THREADS) {
        int prime = *p;
        int btc = bitToClear(exp, k0 + c, prime, *invp);
        // int btc = *btcp;
        while (btc < NBITS) {
          atomicOr(words + (btc >> 5), 1 << (btc & 0x1f));
          btc += prime;
        }
        // *btcp = btc - NBITS;
      }
      // timeSieve = clock() - time0;
    } else {
}

#ifndef NDEBUG
  U3 r2 = expMod2(flushedExp, q);
  if (!(r.a == r2.a && r.b == r2.b && r.c == r2.c)) {
    printf("%08x%08x%08x %08x%08x%08x %08x%08x%08x\n", r.c, r.b, r.a, r2.c, r2.b, r2.a, q.c, q.b, q.a);
  }
  assert(r.a == r2.a && r.b == r2.b && r.c == r2.c);
#endif

  
#ifndef NDEBUG
__device__ U3 mod(U5 x, U3 m) {
  assert(m.c && !(m.c & 0xc0000000));
  int sh = __clz(m.c) + 1;
  if (sh > 26) {
    m = shl(m, sh - 26);
    sh = 26;
  }
  assert(sh >= 3 && sh <= 26);
  u32 R = 0xffffffffffffffffULL / ((0x100000000ULL | shl(m.b, m.c, sh)) + 1);
  u32 n = mulhi(x.e, R);
  U4 t = sub((U4) {x.b, x.c, x.d, x.e}, shl(mul(m, n), sh));
  x = (U5){x.a, t.a, t.b, t.c, t.d};
  assert(!(x.e & 0xfffffff8));
  n = mulhi(shl(x.d, x.e, 29), R);
  U5 mn = shl(_U5(mul(m, n)), sh + 3);
  x = sub(x, mn);
  assert(!x.e && !(x.d & 0xffffffc0));
  n = mulhi(shl(x.c, x.d, 26), R) >> (26 - sh);
  t = sub((U4) {x.a, x.b, x.c, x.d}, mul(m, n));
  assert(!t.d);
  assert(!(t.c >> (35 - sh)));
  return (U3) {t.a, t.b, t.c};
}

__device__ U3 expMod2(u32 exp, U3 m) {
  assert(exp & 0x80000000);
  int sh = exp >> 26;
  assert(sh >= 32 && sh < 64);
  U3 a = mod((U5){0, 0, 0, 0, 1 << (sh - 32)}, m);
  u32 mp = mprime(m.a);
  for (exp <<= 6; exp; exp += exp) {
    a = montRed(square(a), m, mp);
    if (exp & 0x80000000) { a = shl(a, 1); }
  }
  return montRed(_U6(a), m, mp);
}
#endif
*/
