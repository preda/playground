#include "hip/hip_runtime.h"
// Copyright (c) Mihai Preda, 2015 - 2016

/*
  Aa program for trial factoring of Mersenne numbers on a CUDA GPU.

  Mersenne numbers are of the form 2**exp - 1; see http://www.mersenne.org/various/math.php
  This is inpired by mfaktc: http://www.mersenneforum.org/mfaktc/

  For a given mersenne number 2**exp-1, where exp is prime, the factors are of the form
  m = 2*k*exp + 1, and we're interested only in prime factors.

  Limits: exp < 2**31; 2**64 < m < 2**76.

  First prime candidate factors are generated -- this is called "sieving" because it uses
  Erathostene's sieve. Next each candidate m is tested by the computing the modular
  exponentiation reminder r = 2**exp modulo m. If this reminder is equal to 1, it means
  that m is a factor of 2^exp-1, and thus the mersenne number is not prime.

  
  Naming conventions used:

  1. type names:
     - u8, u16, u32, u64, u128: unsigned integer with the given number of *bits*.
     - U2, U3, U4, etc: unsigned long integer with the given number of 32-bit words.
       The words of a long integer are named "a", "b", "c", etc, a being the least-significant.
       
  2. operators on long integers:
     - usual: +, -, *.
     - bit shifts: <<, >>.
     - shr1w(): word shift right
     - funnel shift returning one word: shl, shr
     - cast to larger type, e.g. _U4(U3 x)
     - mulLow(): multiplication computing only the lower words
     - shr3wMul(): multiplication computing  only the higher words
     - equality ==
     - square
*/

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <sys/unistd.h>

#define DEVICE __device__ static
#define ASIZE(a) (sizeof(a) / sizeof(a[0]))

#include "widemath.h"
#include "debug.h"

// Table of small primes.
DEVICE const u32 primes[] = {
#include "primes-1M.inc"
};
// Number of pre-computed primes for sieving.
#define NPRIMES (ASIZE(primes))

// Unit tests. A series of pairs (exponent, k) where k represents a factor.
struct Test { u32 exp; u64 k; };
#include "tests.inc"

// Threads for initBtcTabs()
#define INIT_BTC_THREADS 256
// Threads per sieving block.
#define SIEVE_THREADS 512
// Threads per testing block.
#define TEST_THREADS 512

// How many words of shared memory to use for sieving.
#define NWORDS (8 * 1024)
// Bits for sieving (each word is 32 bits).
#define NBITS (NWORDS << 5)
// How many rows are needed at most in a testing block of TEST_THREADS colums.
#define TEST_ROWS (NBITS / 5 / TEST_THREADS + 1)

// Must update acceptClass() when changing these.
#define NCLASS     (4 * 3 * 5 * 7 * 11)
// Out of NCLASS, how many classes pass acceptClass(). Sync with NCLASS.
#define NGOODCLASS (2 * 2 * 4 * 6 * 10)

// Some powers of 2 as floats, used by inv160()
#define TWO16f  65536.0f
#define TWO17f  131072.0f
#define TWO28f  268435456.0f
#define TWO32f  4294967296.0f
#define TWO64f  18446744073709551616.0f

// Helper to check and bail out on any CUDA error.
#define CUDA_CHECK  {hipError_t _err = hipGetLastError(); if (_err) { printf("CUDA error: %s\n", hipGetErrorString(_err)); return 0; }}

inline void checkCuda(hipError_t result) {
  if (result != hipSuccess) { printf("CUDA Runtime Error: %s\n", hipGetErrorString(result)); }
}

// Returns whether 2 * c * exp + 1 is 1 or 7 modulo 8.
// Any Marsenne factor must be of this form. See http://www.mersenne.org/various/math.php
bool q1or7mod8(u32 exp, u32 c) { return !(c & 3) || ((c & 3) + (exp & 3) == 4); }

// whether 2 * c * exp + 1 != 0 modulo prime
bool notMultiple(u32 exp, u32 c, unsigned prime) { return (2 * c * (u64) exp) % prime != prime - 1; }
// { return (2 * c * (u64) exp + 1) % prime; }

bool acceptClass(u32 exp, u32 c) {
#define P(p) notMultiple(exp, c, p)
  return q1or7mod8(exp, c) && P(3) && P(5) && P(7) && P(11);
#undef P
}

u64 timeMillis() {
  struct timeval tv;
  gettimeofday(&tv, 0);
  return tv.tv_sec * 1000 + tv.tv_usec / 1000;
}

// Table with inv(exp). Initialized once per exponent.
DEVICE u32 invTab[NPRIMES];

// "Bit to clear" table, depends on exponent, k0, and class; initialized once per exponent.
DEVICE int btcTabs[NGOODCLASS][NPRIMES];

// Sieved bits are aggregated from shared memory after sieve() to this global memory block.
DEVICE u32 sievedBits[NGOODCLASS][NWORDS];

// Deltas of Ks for testing. This is a derivate of the sieved bits.
DEVICE u16 kDeltas[NGOODCLASS][TEST_ROWS * TEST_THREADS];

// Alternative tab, see sieve()
DEVICE u32 kTab[NGOODCLASS][NBITS / 5];
// kTabSize[i] has the number of elements in kTab[i]
DEVICE u32 kTabSize[NGOODCLASS];

__managed__ U3 foundFactor; // If a factor m is found, save it here.
__managed__ u64 foundK; //K for a found factor.
__managed__ int classTab[NGOODCLASS];

// Returns x % m, given u the "inverse" of m (2**160 / m); m at most 77 bits.
DEVICE U3 mod(U5 x, U3 m, U3 u) {
  return (U3){x.a, x.b, x.c} - mulLow(m, shr3wMul((U3) {x.c, x.d, x.e}, u));
}

// float lower approximation of 2**32 / x
DEVICE float floatInv(U3 x) { return __frcp_rd(__ull2float_ru(_u64(shr1w(x)) + 1)); }

// float lower approximation of a + b * 2**32; (__fmaf_rz(b, TWO32f, a))
DEVICE float floatOf(u32 a, u32 b) { return __ull2float_rz(_u64((U2) {a, b})); }

// float lower approximation of (a + b * 2**32) * nf
DEVICE float floatOf(u32 a, u32 b, float nf) { return __fmul_rz(floatOf(a, b), nf); }

// Returns 2**160 / n
DEVICE U3 inv160(U3 n, float nf) {
  // 1
  assert(nf * TWO64f < TWO32f);
  u32 rc = (u32) __fmul_rz(TWO64f, nf);
  U4 q = shl1w((~mulLow(n, rc)) + 1);

  // 2
  float qf = floatOf(q.c, q.d, nf) * TWO16f;
  assert(qf < TWO28f);
  u32 qi = (u32) qf;
  u32 rb = (qi << 16);
  rc += (qi >> 16);
  q = q - ((n * qi) << 16);
  assert(q.d == 0);

  // 3
  qf = floatOf(q.b, q.c, nf);
  assert(qf < (1 << 24));
  qi = (u32) qf;
  U2 rup = (U2){rb, rc} + qi;
  q = q - n * qi;
  assert(q.d == 0);
  
  // 4
  qf = floatOf(q.b, q.c, nf) * TWO17f;
  assert(qf < (1 << 22));
  qi = (u32) qf;
  rup = rup + (qi >> 17);
  U3 ret = (U3) {(qi << 15), rup.a, rup.b};

  // p("n ", n); p("q ", q);
  q = ((U4) {0, q.a, q.b, q.c}) - ((n * qi) << 15);
  // if (q.d) { printf("qi %d qf %.2f %.10f %.10f %.10f %f nf %f", qi, qf, t1, t2, t3, TWO32f, (nf * TWO64f)); p("q4 ", q); }
  assert(q.d == 0);
  
  // 5
  qf = floatOf(q.b, q.c, nf);
  assert(qf < (1 << 20));
  return ret + (u32) qf;
}

// Returns 2**exp % m
DEVICE U3 expMod(u32 exp, U3 m) {
  assert(exp & 0x80000000);
  assert(m.c && !(m.c & 0xffffc000));
  int sh = exp >> 25;
  assert(sh >= 64 && sh < 128);
  exp <<= 7;

  float nf = floatInv(m);
  U3 u = inv160(m, nf);
  U3 a = mod((U5){0, 0, 1 << (sh - 64), 1 << (sh - 96), 0}, m, u);
  do {
    a = mod(square(a), m, u);
    if (exp & 0x80000000) { a <<= 1; }
  } while (exp += exp);
  a = a - mulLow(m, (u32) floatOf(a.b, a.c, nf));
  return (a.c >= m.c && a == (m + 1)) ? (U3) {1, 0, 0} : a;
}

DEVICE u32 modInv32(u64 step, u32 prime) {
  int n = step % prime;
  int q = prime / n;
  int d = prime - q * n;
  int x = -q;
  int prevX = 1;
  while (d) {
    q = n / d;
    { int save = d; d = n - q * d; n = save; }            // n = set(d, n - q * d);
    { int save = x; x = prevX - q * x; prevX = save; }    // prevX = set(x, prevX - q * x);
  }
  return (prevX >= 0) ? prevX : (prevX + prime);
}

// 3 times 64bit modulo, expensive!
DEVICE int bitToClear(u32 exp, u64 k, u32 prime, u32 inv) {
  u32 kmod = k % prime;
  u32 qmod = (kmod * (u64) (exp << 1) + 1) % prime;
  return (prime - qmod) * (u64) inv % prime;
}

__global__ void __launch_bounds__(1024) initInvTab(u32 exp) {
  assert(gridDim.x * blockDim.x == NPRIMES);
  u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  invTab[id] = modInv32(2 * NCLASS * (u64) exp, primes[id]);
}

__global__ void __launch_bounds__(INIT_BTC_THREADS) initBtcTabs(u32 exp, u64 kBase) {
  assert(gridDim.x == NGOODCLASS);
  int *btcTab = btcTabs[blockIdx.x];
  u64 k = kBase + classTab[blockIdx.x];
  // if (!threadIdx.x) { printf("start class %d (%d)\n", classTab[blockIdx.x], blockIdx.x); }
  for (int id = threadIdx.x; id < NPRIMES; id += blockDim.x) {
    btcTab[id] = bitToClear(exp, k, primes[id], invTab[id]);
  }
  // if (!threadIdx.x) { printf("ended class %d (%d)\n", classTab[blockIdx.x], blockIdx.x); }
}

__global__ void __launch_bounds__(TEST_THREADS) test(u32 doubleExp, u32 flushedExp, u64 k0) {
  // if (!threadIdx.x) { printf("Start %d\n", blockIdx.x); }
  U3 m0 = _U2(k0 + classTab[blockIdx.x]) * doubleExp;
  m0.a |= 1;
  int n = 0;
  u32 *kTabBase = kTab[blockIdx.x];
  for (u32 *p = kTabBase + threadIdx.x, *end = kTabBase + kTabSize[blockIdx.x]; p < end; p += blockDim.x) {
    u32 delta = *p;
    U3 r = expMod(flushedExp, m0 + _U2(NCLASS * (u64) delta * doubleExp));
    ++n;
    if (r == (U3) {1, 0, 0}) {
      foundK = k0 + classTab[blockIdx.x] + NCLASS * (u64) delta;
      printf("factor k: %llu\n", foundK);
    }
  }
  // if (!threadIdx.x) { printf("End %d %d\n", blockIdx.x, n); }
}

__global__ void testSingle(u32 doubleExp, u32 flushedExp, u64 k) {
  U3 m = _U2(k) * doubleExp;
  m.a |= 1;
  U3 r = expMod(flushedExp, m);
  foundFactor = r;
}

// Returns the position of the most significant bit that is set.
DEVICE int bfind(u32 x) { int r; asm("bfind.u32 %0, %1;": "=r"(r): "r"(x)); return r; }

// Sieve bits using shared memory.
// For each prime from the primes[] table, starting at a position corresponding to a
// multiple of prime ("btc"), periodically set the bit to indicate a non-prime.
__global__ void __launch_bounds__(SIEVE_THREADS) sieve() {
  __shared__ u32 words[NWORDS];

  // Set shared memory to zero.
  for (int i = threadIdx.x; i < NWORDS; i += blockDim.x) { words[i] = 0; }
  __syncthreads();

  // Sieve bits.
  int *btcTab = btcTabs[blockIdx.x];
  for (int i = threadIdx.x; i < NPRIMES; i += blockDim.x) {
    int prime = primes[i];
    int btc = btcTab[i];
    while (btc < NBITS) {
      atomicOr(words + (btc >> 5), 1 << (btc & 0x1f));
      btc += prime;
    }
    btcTab[i] = btc - NBITS;
  }
  __syncthreads();

  u32 bits = ~words[threadIdx.x];
  words[threadIdx.x] = 0;
  __syncthreads();
  
  int popc = __popc(bits);
  for (int i = blockDim.x + threadIdx.x; i < NWORDS; i += blockDim.x) { popc += __popc(~words[i]); }
  u32 *out = kTab[blockIdx.x] + atomicAdd(words, popc);
  __syncthreads();

  if (threadIdx.x == 0) { kTabSize[blockIdx.x] = words[0]; }
  int i = threadIdx.x;
  while (true) {
    while (bits) {
      int bit = __clz(__brev(bits)); // Equivalent to: __ffs(bits) - 1; 
      bits &= bits - 1;  // Equivalent to: bits &= ~(1 << bit); but likely faster
      
      // int bit = bfind(bits);
      // bits &= ~(1 << bit);
      *out++ = (i << 5) + bit;
    }
    if ((i += blockDim.x) >= NWORDS) { break; }
    bits = ~words[i];
  }
}

  /*
  // Copy shared memory to global memory.
  u32 *out = sievedBits[blockIdx.x];
  for (int i = threadIdx.x; i < NWORDS; i += blockDim.x) {
    out[i] = ~words[i];
  }
  */

// Among all the NCLASS classes, select the ones that are "good",
// i.e. not corresponding to a multiple of a small prime.
void initClasses(u32 exp) {
  int nClass = 0;
  for (int c = 0; c < NCLASS; ++c) {
    if (acceptClass(exp, c)) {
      classTab[nClass++] = c;
      // if (c == 992) { printf("class id %d\n", nClass - 1); }
    }
  }
  assert(nClass == NGOODCLASS);
}

// The smallest k that produces a factor m = (2*k*exp + 1) such that m >= 2**bits
u64 calculateK(u32 exp, int bits) { return ((((u128) 1) << (bits - 1)) + (exp - 2)) / exp; }

// Run one unit-test case.
bool testOne(u32 exp, u64 k) {  
  u32 flushedExp = exp << __builtin_clz(exp);
  u32 doubleExp = exp + exp;
  printf("\r%10u %20llu", exp, k);
  testSingle<<<1, 1>>>(doubleExp, flushedExp, k);
  hipDeviceSynchronize(); CUDA_CHECK;
  if (foundFactor.a != 1 || foundFactor.b || foundFactor.c) {
    printf("ERROR %10u %20llu m ", exp, k);
    return false;
  }
  return true;
}

void extractBits(u64 *bits, u16 (*deltas)[TEST_ROWS * TEST_THREADS]) {
  u32 prev[TEST_THREADS];
  u32 *prevEnd = prev + TEST_THREADS;
  
  u64 *p = bits;
  for (int ci = 0; ci < NGOODCLASS; ++ci) {
    u16 *deltap = deltas[ci];
    u32 *prevp  = prev;

    memset(prev, 0, sizeof(prev));
    u32 currentWordPos = 0;

    for (u64 *end = p + (NWORDS/2); p < end; ++p) {
      u64 w = *p;
      while (w) {
        u32 bit = currentWordPos + __builtin_ctzl(w);
        w &= (w - 1);
        *deltap++ = (u16) (bit - *prevp);
        *prevp++ = bit;
        if (prevp == prevEnd) { prevp = prev; }
      }
      currentWordPos += 64;
    }
    assert(deltap + TEST_THREADS <= deltas[ci + 1]);
    memset(deltap, 0xff, sizeof(u16) * TEST_THREADS);
  }
}

void time(const char *s = 0) {
  static u64 prev = 0;
  u64 now = timeMillis();
  if (prev && s) {
    printf("%s: %llu ms\n", s, now - prev);
  }
  prev = now;
}

int main(int argc, char **argv) {
  assert(argc > 0);
  assert(NPRIMES % 1024 == 0);
  
  // hipSetDevice(1);
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync); CUDA_CHECK;
  
  if (argc == 1) {
    printf("Running selftest..\n");
    for (Test *t = tests, *end = tests + ASIZE(tests); t < end; ++t) {
      if (!testOne(t->exp, t->k)) { return -1; }
    }
    printf("\n%lu tests passed ok\n", ASIZE(tests));
    return 0;
  }

  u32 exp = (u32) atol(argv[1]);
  int startPow2 = (argc >= 3) ? atoi(argv[2]) : 65;
  
  time();
  initClasses(exp);
  time("initClasses");

  u64 k0 = calculateK(exp, startPow2);
  k0 -= k0 % NCLASS;
  u64 kEnd = calculateK(exp, startPow2 + 1);
  kEnd += (NCLASS - (kEnd % NCLASS)) % NCLASS;
  u32 kStep = NBITS * NCLASS;
  u32 repeat = (kEnd - k0 + (kStep - 1)) / kStep;
  u32 flushedExp = exp << __builtin_clz(exp);
  u32 doubleExp = exp + exp;
  printf("k range: %llu - %llu. %u iterations.\n", k0, kEnd, repeat);
    
  initInvTab<<<NPRIMES/1024, 1024>>>(exp); CUDA_CHECK;
  // hipDeviceSynchronize(); time("initInvTab");
  initBtcTabs<<<NGOODCLASS, INIT_BTC_THREADS>>>(exp, k0); CUDA_CHECK;
  // hipDeviceSynchronize(); time("initBtcTabs");

  u64 *hostBits = 0;
  checkCuda(hipHostAlloc(&hostBits, NGOODCLASS * NWORDS * 4, 0));
  time("Alloc hostBits");

  u16 (*deltas)[TEST_ROWS * TEST_THREADS];
  checkCuda(hipHostAlloc(&deltas, NGOODCLASS * sizeof(deltas[0]), 0));
  time("Alloc deltas");
  hipDeviceSynchronize(); time("init inv + btc");

  /*
  sieve<<<NGOODCLASS, SIEVE_THREADS>>>();
  hipDeviceSynchronize(); CUDA_CHECK; time("Sieve");
  */
  
  for (int i = 0; i < repeat; ++i, k0 += kStep) {
    sieve<<<NGOODCLASS, SIEVE_THREADS>>>();
    hipDeviceSynchronize(); CUDA_CHECK;
    // time("Sieve");

    /*
    hipMemcpyFromSymbol(hostBits, HIP_SYMBOL(sievedBits), NGOODCLASS * NWORDS * 4, 0, hipMemcpyDeviceToHost);
    CUDA_CHECK; time("Copy from device");
  
    extractBits(hostBits, deltas); time("Extract bits");

    hipMemcpyToSymbol(HIP_SYMBOL(kDeltas), deltas, NGOODCLASS * sizeof(deltas[0]));
    CUDA_CHECK; time("Copy to device");
    */

    test<<<NGOODCLASS, TEST_THREADS>>>(doubleExp, flushedExp, k0);
    hipDeviceSynchronize(); CUDA_CHECK;
    // time("Test");
    char buf[64];
    snprintf(buf, sizeof(buf), "cycle %4d", i);
    time(buf);
    
    if (foundK) {
      printf("*** K *** %lld\n", foundK);
      break;
    }
  }
  hipDeviceSynchronize();
}

  /*
  int p1=-1, p2=-1;
  hipDeviceGetStreamPriorityRange(&p1, &p2);
  CUDA_CHECK;
  printf("Priority %d %d\n", p1, p2);
  
  hipStream_t sieveStream, testStream;
  hipStreamCreateWithPriority(&sieveStream, hipStreamNonBlocking, 0);
  CUDA_CHECK;
  hipStreamCreateWithPriority(&testStream, hipStreamNonBlocking, 1);
  CUDA_CHECK;
  */
